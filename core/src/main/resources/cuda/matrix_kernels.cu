#include "hip/hip_runtime.h"
#include <stdio.h>

#define MAKE_NAME(prefix, fun, T) prefix ## _ ## fun ## _ ## T

#define MAP_FUN_1(fun, T) \
extern "C" \
__global__ void MAKE_NAME(map, fun, T) (int rows, int cols,\
    T *out, int outMajorStride,\
    const T *in, int inMajorStride) {\
  for(int col = threadIdx.x + blockIdx.x * blockDim.x; col < cols; col += blockDim.x * gridDim.x) {\
    for(int row = threadIdx.y + blockIdx.y * blockDim.y; row < rows;  row += blockDim.y * gridDim.y) {\
        out[col * outMajorStride + row] = fun(in[col * inMajorStride + row]);\
    }\
  }\
}

#define MAP_BLOCK_SIZE 32


#define MAP_FUN_2(fun, T) \
extern "C" \
__global__ void MAKE_NAME(map2, fun, T) (int rows, int cols,\
    T *out, int outMajorStride,\
    const T *a, int aMajorStride,\
    const T *b, int bMajorStride) {\
  for(int col = threadIdx.x + blockIdx.x * blockDim.x; col < cols; col += blockDim.x * gridDim.x) {\
    for(int row = threadIdx.y + blockIdx.y * blockDim.y; row < rows;  row += blockDim.y * gridDim.y) {\
        out[col * outMajorStride + row] = fun(a[col * aMajorStride + row], b[col * bMajorStride + row]);\
    }\
  }\
}\
\
extern "C" \
__global__ void MAKE_NAME(map2_v_s, fun, T) (int rows, int cols,\
    T *out, int outMajorStride,\
    const T *a, int aMajorStride,\
    const T b) {\
  for(int col = threadIdx.x + blockIdx.x * blockDim.x; col < cols; col += blockDim.x * gridDim.x) {\
    for(int row = threadIdx.y + blockIdx.y * blockDim.y; row < rows;  row += blockDim.y * gridDim.y) {\
        out[col * outMajorStride + row] = fun(a[col * aMajorStride + row], b);\
    }\
  }\
}\
\
extern "C" \
__global__ void MAKE_NAME(map2_s_v, fun, T) (int rows, int cols,\
    T *out, int outMajorStride,\
    const T a,\
    const T *b, int bMajorStride) {\
  for(int col = threadIdx.x + blockIdx.x * blockDim.x; col < cols; col += blockDim.x * gridDim.x) {\
    for(int row = threadIdx.y + blockIdx.y * blockDim.y; row < rows;  row += blockDim.y * gridDim.y) {\
        out[col * outMajorStride + row] = fun(a, b[col * bMajorStride + row]);\
    }\
  }\
}\
extern "C" \
__global__ void MAKE_NAME(map2_transpose, fun, T) (int rows, int cols,\
    T *out, int outMajorStride,\
    const T *a, int aMajorStride,\
    const T *b, int bMajorStride) {\
\
    int numGroupsX = blockDim.x * gridDim.x;\
  int numGroupsY = blockDim.y * gridDim.y;\
  int firstBlockX = blockDim.x * blockIdx.x;\
  int firstBlockY = blockDim.y * blockIdx.y;\
  __shared__ T tile[MAP_BLOCK_SIZE][MAP_BLOCK_SIZE+1];\
  \
   /*x is row in a, col in b*/\
   /*y is col in a, row in b*/\
  \
  for (int yb = firstBlockY; yb < cols; yb += numGroupsY) {\
    for (int xb = firstBlockX; xb < rows; xb += numGroupsX) {\
       int ylim = min(cols, yb + MAP_BLOCK_SIZE);\
      int xlim = min(rows, xb + MAP_BLOCK_SIZE);\
      \
      \
      /* use threadid.y for x here so that the y loop is on the first blockDim, which
       means coalesced reads*/\
      for (int x = threadIdx.y + xb; x < xlim; x += blockDim.y) {\
        for(int y = threadIdx.x + yb; y < ylim; y += blockDim.x) {\
          tile[x-xb][y-yb] = b[x*bMajorStride + y];\
        }\
      }\
      \
    __syncthreads();\
      for(int y = threadIdx.y + yb; y < ylim; y += blockDim.y) {\
        for (int x = threadIdx.x + xb; x < xlim; x += blockDim.x) {\
          out[x + y*outMajorStride] = fun(a[x + y * aMajorStride], tile[x-xb][y-yb]);\
        }\
      }\
    __syncthreads();\
    }\
  }\
}


/*
  for(int col = threadIdx.x + blockIdx.x * blockDim.x; col < cols; col += blockDim.x * gridDim.x) {\
    for (int j = 0; j < ; j += BLOCK_ROWS)
     block[(threadIdx.y+j)*TILE_DIM + threadIdx.x] = idata[(y+j)*width + x];

    for(int row = threadIdx.y + blockIdx.y * blockDim.y; row < rows;  row += blockDim.y * gridDim.y) {\
        out[col * outMajorStride + row] = fun(a[col * aMajorStride + row], b[col * bMajorStride + row]);\
    }\
  }\
}\
\
*/



 static __inline__ __device__ double shfl_down(double var, int delta, int width=warpSize)
{
    int hi, lo;
    asm volatile( "mov.b64 { %0, %1 }, %2;" : "=r"(lo), "=r"(hi) : "d"(var) );
    hi = __shfl_down( hi, delta, width );
    lo = __shfl_down( lo, delta, width );
    return __hiloint2double( hi, lo );
}

static __inline__ __device__ int shfl_down(int var, int delta, int width=warpSize)
{
    return __shfl_down(var, delta, width);
}

static __inline__ __device__ unsigned int shfl_down(unsigned int var, int delta, int width=warpSize)
{
    int x = __shfl_down(*(int*)&var, delta, width);
    return *(unsigned int*)(&x);
}

static __inline__ __device__ float shfl_down(float var, int delta, int width=warpSize)
{
    return __shfl_down(var, delta, width);
}

#define laneId (threadIdx.x & 0x1f)



#define REDUCE_FUN(fun, T, identity) \
/* Each column gets 1 block of threads. TODO currently blocksize must be 1 warp*/\
extern "C" \
__global__ void MAKE_NAME(reduce, fun, T) (int rows, int cols,\
    T *out,\
    const T *in, int inMajorStride) {\
  /*__shared__ T buffer[32];\*/\
\
  T sum = identity;\
  for(int col = threadIdx.y + blockIdx.y * blockDim.y; col < cols; col += blockDim.y * gridDim.y) {\
    for(int row = threadIdx.x + blockIdx.x * blockDim.x; row < rows;  row += blockDim.x * gridDim.x) {\
        sum = fun(sum, in[col * inMajorStride + row]);\
    }\
  }\
  \
  __syncthreads();\
  for (int i = 1; i < blockDim.x; i *= 2) {\
    T x = shfl_down(sum, i);\
    sum = fun(sum, x);\
  }\
  \
  if(laneId == 0) {\
    out[blockIdx.x * gridDim.y + blockIdx.y] = sum;\
  }\
}\
\
/* Each column gets 1 block of threads. TODO currently blocksize must be 1 warp*/\
extern "C" \
__global__ void MAKE_NAME(reduce_col, fun, T) (int rows, int cols,\
    T *out,\
    const T *in, int inMajorStride) {\
  /*__shared__ T buffer[32];\*/\
\
  for(int col = threadIdx.y + blockIdx.x * blockDim.y; col < cols; col += blockDim.y * gridDim.x) {\
    T sum = identity;\
    for(int row = threadIdx.x; row < rows; row += blockDim.x) {\
      sum = fun(sum, in[col * inMajorStride + row]);\
    }\
    \
    __syncthreads();\
    for (int i = 1; i < blockDim.x; i *= 2) {\
      T x = shfl_down(sum, i);\
      sum = fun(sum, x);\
    }\
    \
    if(laneId == 0) {\
      out[col] = sum;\
    }\
  }\
}\
\
\
/*Each row has its own thread. We should make multiple threads per row, but later. TODO */\
extern "C" \
__global__ void MAKE_NAME(reduce_row, fun, T) (int rows, int cols,\
    T *out,\
    const T *in, int inMajorStride) {\
 /* __shared__ T buffer[32];*/\
\
  int numReducers = blockDim.x * gridDim.x;\
  for(int row = threadIdx.x + blockIdx.x * blockDim.x; row < rows; row += numReducers) {\
    T sum = identity;\
    for(int col = 0; col < cols; col++) {\
      sum = fun(sum, in[col * inMajorStride + row]);\
    }\
    \
    out[row] = sum;\
  }\
}\
        

#include "function_decls.cuh"

