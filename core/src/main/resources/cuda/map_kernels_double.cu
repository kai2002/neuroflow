
#include <hip/hip_runtime.h>

extern "C"
__global__ void vec_set (size_t n, double *result, double  value)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = value;
    }
}


//=== Vector arithmetic ======================================================

extern "C"
__global__ void vec_add (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x[id] + y[id];
    }
}


extern "C"
__global__ void vec_sub (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x[id] - y[id];
    }
}


extern "C"
__global__ void vec_mul (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x[id] * y[id];
    }
}


extern "C"
__global__ void vec_div (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x[id] / y[id];
    }
}

extern "C"
__global__ void vec_negate (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = -x[id];
    }
}




//=== Vector-and-scalar arithmetic ===========================================

extern "C"
__global__ void vec_addScalar (size_t n, double *result, double  *x, double  y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x[id] + y;
    }
}


extern "C"
__global__ void vec_subScalar (size_t n, double *result, double  *x, double  y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x[id] - y;
    }
}


extern "C"
__global__ void vec_mulScalar (size_t n, double *result, double  *x, double  y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x[id] * y;
    }
}


extern "C"
__global__ void vec_divScalar (size_t n, double *result, double  *x, double  y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x[id] / y;
    }
}




extern "C"
__global__ void vec_scalarAdd (size_t n, double *result, double  x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x + y[id];
    }
}


extern "C"
__global__ void vec_scalarSub (size_t n, double *result, double  x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x - y[id];
    }
}


extern "C"
__global__ void vec_scalarMul (size_t n, double *result, double  x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x * y[id];
    }
}


extern "C"
__global__ void vec_scalarDiv (size_t n, double *result, double  x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x / y[id];
    }
}









//=== Vector comparison ======================================================

extern "C"
__global__ void vec_lt (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] < y[id])?1.0f:0.0f;
    }
}


extern "C"
__global__ void vec_lte (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] <= y[id])?1.0f:0.0f;
    }
}


extern "C"
__global__ void vec_eq (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] == y[id])?1.0f:0.0f;
    }
}


extern "C"
__global__ void vec_gte (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] >= y[id])?1.0f:0.0f;
    }
}


extern "C"
__global__ void vec_gt (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] > y[id])?1.0f:0.0f;
    }
}



extern "C"
__global__ void vec_ne (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] != y[id])?1.0f:0.0f;
    }
}




//=== Vector-and-scalar comparison ===========================================

extern "C"
__global__ void vec_ltScalar (size_t n, double *result, double  *x, double  y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] < y)?1.0f:0.0f;
    }
}


extern "C"
__global__ void vec_lteScalar (size_t n, double *result, double  *x, double  y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] <= y)?1.0f:0.0f;
    }
}


extern "C"
__global__ void vec_eqScalar (size_t n, double *result, double  *x, double  y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] == y)?1.0f:0.0f;
    }
}


extern "C"
__global__ void vec_gteScalar (size_t n, double *result, double  *x, double  y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] >= y)?1.0f:0.0f;
    }
}


extern "C"
__global__ void vec_gtScalar (size_t n, double *result, double  *x, double  y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] > y)?1.0f:0.0f;
    }
}


extern "C"
__global__ void vec_neScalar (size_t n, double *result, double  *x, double  y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] != y)?1.0f:0.0f;
    }
}











//=== Vector math (one argument) =============================================


// Calculate the arc cosine of the input argument.
extern "C"
__global__ void vec_acos (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = acosf(x[id]);
    }
}


// Calculate the nonnegative arc hyperbolic cosine of the input argument.
extern "C"
__global__ void vec_acosh (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = acoshf(x[id]);
    }
}


// Calculate the arc sine of the input argument.
extern "C"
__global__ void vec_asin (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = asinf(x[id]);
    }
}


// Calculate the arc hyperbolic sine of the input argument.
extern "C"
__global__ void vec_asinh (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = asinhf(x[id]);
    }
}


// Calculate the arc tangent of the input argument.
extern "C"
__global__ void vec_atan (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = atanf(x[id]);
    }
}


// Calculate the arc hyperbolic tangent of the input argument.
extern "C"
__global__ void vec_atanh (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = atanhf(x[id]);
    }
}


// Calculate the cube root of the input argument.
extern "C"
__global__ void vec_cbrt (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = cbrtf(x[id]);
    }
}


// Calculate ceiling of the input argument.
extern "C"
__global__ void vec_ceil (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = ceilf(x[id]);
    }
}


// Calculate the cosine of the input argument.
extern "C"
__global__ void vec_cos (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = cosf(x[id]);
    }
}


// Calculate the hyperbolic cosine of the input argument.
extern "C"
__global__ void vec_cosh (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = coshf(x[id]);
    }
}


// Calculate the cosine of the input argument � p .
extern "C"
__global__ void vec_cospi (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = cospif(x[id]);
    }
}


// Calculate the complementary error function of the input argument.
extern "C"
__global__ void vec_erfc (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = erfcf(x[id]);
    }
}


// Calculate the inverse complementary error function of the input argument.
extern "C"
__global__ void vec_erfcinv (size_t n, double *result, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = erfcinvf(y[id]);
    }
}


// Calculate the scaled complementary error function of the input argument.
extern "C"
__global__ void vec_erfcx (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = erfcxf(x[id]);
    }
}


// Calculate the error function of the input argument.
extern "C"
__global__ void vec_erf (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = erff(x[id]);
    }
}


// Calculate the inverse error function of the input argument.
extern "C"
__global__ void vec_erfinv (size_t n, double *result, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = erfinvf(y[id]);
    }
}


// Calculate the base 10 exponential of the input argument.
extern "C"
__global__ void vec_exp10 (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = exp10f(x[id]);
    }
}


// Calculate the base 2 exponential of the input argument.
extern "C"
__global__ void vec_exp2 (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = exp2f(x[id]);
    }
}


// Calculate the base e exponential of the input argument.
extern "C"
__global__ void vec_exp (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = expf(x[id]);
    }
}


// Calculate the base e exponential of the input argument, minus 1.
extern "C"
__global__ void vec_expm1 (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = expm1f(x[id]);
    }
}


// Calculate the absolute value of its argument.
extern "C"
__global__ void vec_fabs (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = fabsf(x[id]);
    }
}


// Calculate the largest integer less than or equal to x.
extern "C"
__global__ void vec_floor (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = floorf(x[id]);
    }
}


// Calculate the value of the Bessel function of the first kind of order 0 for the input argument.
extern "C"
__global__ void vec_j0 (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = j0f(x[id]);
    }
}


// Calculate the value of the Bessel function of the first kind of order 1 for the input argument.
extern "C"
__global__ void vec_j1 (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = j1f(x[id]);
    }
}


// Calculate the natural logarithm of the absolute value of the gamma function of the input argument.
extern "C"
__global__ void vec_lgamma (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = lgammaf(x[id]);
    }
}


// Calculate the base 10 logarithm of the input argument.
extern "C"
__global__ void vec_log10 (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = log10f(x[id]);
    }
}


// Calculate the value of l o g e ( 1 + x ) .
extern "C"
__global__ void vec_log1p (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = log1pf(x[id]);
    }
}


// Calculate the base 2 logarithm of the input argument.
extern "C"
__global__ void vec_log2 (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = log2f(x[id]);
    }
}


// Calculate the doubleing point representation of the exponent of the input argument.
extern "C"
__global__ void vec_logb (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = logbf(x[id]);
    }
}


// Calculate the natural logarithm of the input argument.
extern "C"
__global__ void vec_log (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = logf(x[id]);
    }
}


// Calculate the standard normal cumulative distribution function.
extern "C"
__global__ void vec_normcdf (size_t n, double *result, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = normcdff(y[id]);
    }
}


// Calculate the inverse of the standard normal cumulative distribution function.
extern "C"
__global__ void vec_normcdfinv (size_t n, double *result, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = normcdfinvf(y[id]);
    }
}


// Calculate reciprocal cube root function.
extern "C"
__global__ void vec_rcbrt (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = rcbrtf(x[id]);
    }
}


// Round input to nearest integer value in doubleing-point.
extern "C"
__global__ void vec_rint (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = rintf(x[id]);
    }
}


// Round to nearest integer value in doubleing-point.
extern "C"
__global__ void vec_round (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = roundf(x[id]);
    }
}


// Calculate the reciprocal of the square root of the input argument.
extern "C"
__global__ void vec_rsqrt (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = rsqrtf(x[id]);
    }
}


// Calculate the sine of the input argument.
extern "C"
__global__ void vec_sin (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = sinf(x[id]);
    }
}


// Calculate the hyperbolic sine of the input argument.
extern "C"
__global__ void vec_sinh (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = sinhf(x[id]);
    }
}


// Calculate the sine of the input argument � p .
extern "C"
__global__ void vec_sinpi (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = sinpif(x[id]);
    }
}


// Calculate the square root of the input argument.
extern "C"
__global__ void vec_sqrt (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = sqrtf(x[id]);
    }
}


// Calculate the tangent of the input argument.
extern "C"
__global__ void vec_tan (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = tanf(x[id]);
    }
}


// Calculate the hyperbolic tangent of the input argument.
extern "C"
__global__ void vec_tanh (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = tanhf(x[id]);
    }
}


// Calculate the gamma function of the input argument.
extern "C"
__global__ void vec_tgamma (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = tgammaf(x[id]);
    }
}


// Truncate input argument to the integral part.
extern "C"
__global__ void vec_trunc (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = truncf(x[id]);
    }
}


// Calculate the value of the Bessel function of the second kind of order 0 for the input argument.
extern "C"
__global__ void vec_y0 (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = y0f(x[id]);
    }
}


// Calculate the value of the Bessel function of the second kind of order 1 for the input argument.
extern "C"
__global__ void vec_y1 (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = y1f(x[id]);
    }
}











//=== Vector math (two arguments) ============================================





// Create value with given magnitude, copying sign of second value.
extern "C"
__global__ void vec_copysign (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = copysignf(x[id], y[id]);
    }
}

// Compute the positive difference between x and y.
extern "C"
__global__ void vec_fdim (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = fdimf(x[id], y[id]);
    }
}

// Divide two doubleing point values.
extern "C"
__global__ void vec_fdivide (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = fdividef(x[id], y[id]);
    }
}

// Determine the maximum numeric value of the arguments.
extern "C"
__global__ void vec_fmax (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = fmaxf(x[id], y[id]);
    }
}

// Determine the minimum numeric value of the arguments.
extern "C"
__global__ void vec_fmin (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = fminf(x[id], y[id]);
    }
}

// Calculate the doubleing-point remainder of x / y.
extern "C"
__global__ void vec_fmod (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = fmodf(x[id], y[id]);
    }
}

// Calculate the square root of the sum of squares of two arguments.
extern "C"
__global__ void vec_hypot (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = hypotf(x[id], y[id]);
    }
}

// Return next representable single-precision doubleing-point value afer argument.
extern "C"
__global__ void vec_nextafter (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = nextafterf(x[id], y[id]);
    }
}

// Calculate the value of first argument to the power of second argument.
extern "C"
__global__ void vec_pow (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = powf(x[id], y[id]);
    }
}

// Compute single-precision doubleing-point remainder.
extern "C"
__global__ void vec_remainder (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = remainderf(x[id], y[id]);
    }
}




