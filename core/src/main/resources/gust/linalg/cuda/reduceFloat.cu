
#include <hip/hip_runtime.h>
/*
 * This code is based on the the sample from JCuda.org/Samples, which in turn
 * is based on the NVIDIA 'reduction' CUDA sample,
 * Copyright 1993-2010 NVIDIA Corporation.
 */
extern "C"
__global__ void reduce(float *g_idata, float *g_odata, unsigned int n)
{
    extern __shared__ float sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x*2 + threadIdx.x;
    unsigned int gridSize = blockDim.x*2*gridDim.x;

    float product = 1.0f;

    while (i < n)
    {
        product *= g_idata[i];
        // ensure we don't read out of bounds
        if (i + blockDim.x < n)
            product *= g_idata[i+blockDim.x];
        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = product;
    __syncthreads();


    // do reduction in shared mem
    if (blockDim.x >= 512) { if (tid < 256) { sdata[tid] = product = product * sdata[tid + 256]; } __syncthreads(); }
    if (blockDim.x >= 256) { if (tid < 128) { sdata[tid] = product = product * sdata[tid + 128]; } __syncthreads(); }
    if (blockDim.x >= 128) { if (tid <  64) { sdata[tid] = product = product * sdata[tid +  64]; } __syncthreads(); }

    if (tid < 32)
    {
        volatile float* smem = sdata;
        if (blockDim.x >=  64) { smem[tid] = product = product * smem[tid + 32]; }
        if (blockDim.x >=  32) { smem[tid] = product = product * smem[tid + 16]; }
        if (blockDim.x >=  16) { smem[tid] = product = product * smem[tid +  8]; }
        if (blockDim.x >=   8) { smem[tid] = product = product * smem[tid +  4]; }
        if (blockDim.x >=   4) { smem[tid] = product = product * smem[tid +  2]; }
        if (blockDim.x >=   2) { smem[tid] = product = product * smem[tid +  1]; }
    }

    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = sdata[0];
}