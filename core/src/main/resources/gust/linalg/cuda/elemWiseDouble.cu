
#include <hip/hip_runtime.h>
/*
 * Kernel for calulating the element-wise product of two matrices
 * m, n --> dimensions of matrices A, B, C
 */
extern "C" {
__global__ void hadamard(int m, int n, double *A, int lda, double *B, int ldb, double *C, int ldc)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= m || j >= n) return;

    C[i + j*ldc] = A[i + j*lda] * B[i + j*ldb];
}
}

/*
 * Matrix sum, parameters as above
 */
extern "C" {
 __global__ void matrix_sum(int m, int n, double *A, int lda, double *B, int ldb, double *C, int ldc)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= m || j >= n) return;

    C[i + j*ldc] = A[i + j*lda] + B[i + j*ldb];
}
}

/*
 * Copy that allows us to move around pieces of a matrix
 */
extern "C" {
 __global__ void copy(int m, int n, double *dst, int lddst, double *src, int ldsrc)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= m || j >= n) return;

    dst[i + j*lddst] = src[i + j*ldsrc];
}
}